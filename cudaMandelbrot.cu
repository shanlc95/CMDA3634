#include "hip/hip_runtime.h"
/*

To compile:
nvcc -O3  -o cudaMandelbrot cudaMandelbrot.cu png_util.c -I. -lm -lpng

To run:
./cudaMandelbrot

*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

extern "C"
{
#include "png_util.h"
}

__global__ void mandelbrot(const int NRe, 
			   const int NIm,
			   const float minRe,
			   const float minIm,
			   const float dRe,
			   const float dIm,
			   const float cRe,
			   const float cIm,
			   float* __restrict__ c_count ) {

  int n,m;
  
  n = threadIdx.x + blockIdx.x*blockDim.x;
  m = threadIdx.y + blockIdx.y*blockDim.y;

  float zRe = minRe + n*dRe;
  float zIm = minIm + m*dIm;
 
  int Nt = 200;
  int t, cnt=0;
  for(t=0;t<Nt;++t){
    
    // z = z^2 + c
    //   = (zRe + i*zIm)*(zRe + i*zIm) + (cRe + i*cIm)
    //   = zRe^2 - zIm^2 + 2*i*zIm*zRe + cRe + i*cIm
    float zReTmp = zRe*zRe - zIm*zIm + zRe;
    zIm = 2.f*zIm*zRe + cIm;
    zRe = zReTmp;
    
    cnt += (zRe*zRe+zIm*zIm<4.f);
  }
  
  c_count[n + m*NRe] = cnt;

}


int main(int argc, char **argv){

  const int NRe = 2048;
  const int NIm = 2048;

  const float cRe = 0.285;
  const float cIm = 0.01;

  /* box containing sample points */
  const float centRe = -.759856, centIm= .125547;
  const float diam  = 0.151579;
  const float minRe = centRe-0.5*diam;
  const float remax = centRe+0.5*diam;
  const float minIm = centIm-0.5*diam;
  const float immax = centIm+0.5*diam;

  const float dRe = (remax-minRe)/(NRe-1.f);
  const float dIm = (immax-minIm)/(NIm-1.f);

  float *h_count = (float*) calloc(NRe*NIm, sizeof(float));

  float *c_count;
  
  hipMalloc(&c_count, NRe*NIm*sizeof(float));
  
  // specify two dimensional array of threads
  int TRe = 16;
  int TIm = 16;
  int BRe = NRe/TRe;
  int BIm = NIm/TIm;
  
  dim3 B(BRe, BIm);
  dim3 T(TRe, TIm);
  
  double tic = clock();


  // call mandelbrot from here
  mandelbrot <<< B, T >>> (NRe, NIm, minRe, minIm, dRe, dIm, cRe, cIm, c_count);

  hipDeviceSynchronize();

  double toc = clock();

  double elapsed = (toc-tic)/CLOCKS_PER_SEC;
  
  printf("elapsed time %f\n", elapsed);

  hipMemcpy(h_count, c_count, NRe*NIm*sizeof(float), hipMemcpyDeviceToHost);

  FILE *png = fopen("cudaMandelbrot.png", "w");
  write_hot_png(png, NRe, NIm, h_count, 0, 80);
  fclose(png);

}
